#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <string>
#include <math.h>
#include "TiMatrix.h"
#include "TiMatrixType.h"

using namespace std;

__global__ void GetDiaKernel(ELL_Matrix DevA, float *D);
__global__ void DLMulKernel(ELL_Matrix DevK, ELL_Matrix DevA, float *D);
__global__ void ELLMulKernel(ELL_Matrix DevK, ELL_Matrix DevM, ELL_Matrix DevA);

float ELLMul(string filename);


int main()
{
	float totaltime = 0;
	string filename;

	cout<<"please input matrix market file name:";
	cin>>filename;

	for(int i = 0 ; i < 5 ; i++)
		totaltime += ELLMul(filename);

	cout<<"the average cost time is "<<totaltime/5<<" ms "<<endl;

	return 0;

}



//����device memory������
float ELLMul(string filename)
{
	
	

	CSR_Matrix origin;
	ELL_Matrix A;
	ELL_Matrix K;
	ELL_Matrix M;
	float *D;
	
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	ReadMatrixMarketFile(filename.c_str(),origin);
	cout<<"Reading matrix..."<<endl;

	CSR2ELL(origin, A);

	

	//print the matrix
	cout<<A.nRow<<"    "<<A.nCol<<"    "<<A.nonzeroes<<endl;
	//PrintELLFormat(A);

	K.nRow = A.nRow;
	K.nCol = A.nCol;
	K.nonzeroes = A.nonzeroes;

	K.mData = new float[A.nRow*A.nCol];
	K.mIndex = new int[A.nRow*A.nCol];


	M.nRow = A.nRow;
	M.nonzeroes = A.nonzeroes;
	M.nCol = A.nCol;
	M.mData = new float[A.nRow*A.nCol];
	M.mIndex = new int[A.nRow*A.nCol];

	D = new float[A.nRow];

	ELL_Matrix DevK;
	ELL_Matrix DevM;
	ELL_Matrix DevA;
	float *DevD;


	DevA.nRow = A.nRow;
	DevA.nCol = A.nCol;
	DevA.nonzeroes = A.nonzeroes;

	DevK.nRow = K.nRow;
	DevK.nCol = K.nCol;
	DevK.nonzeroes = K.nonzeroes;

	DevM.nRow = M.nRow;
	DevM.nCol = M.nCol;
	DevM.nonzeroes = M.nonzeroes;


	hipMalloc((void **)&DevK.mData,sizeof(float)*A.nRow*A.nCol);
	hipMalloc((void **)&DevK.mIndex,sizeof(int)*A.nRow*A.nCol);

	hipMalloc((void **)&DevA.mData,sizeof(float)*A.nRow*A.nCol);
	hipMalloc((void **)&DevA.mIndex,sizeof(int)*A.nRow*A.nCol);

	hipMalloc((void **)&DevD, sizeof(float)*A.nRow);


	hipMemcpy(DevA.mData,A.mData,sizeof(float)*A.nRow*A.nCol,hipMemcpyHostToDevice);
	hipMemcpy(DevA.mIndex,A.mIndex,sizeof(int)*A.nRow*A.nCol,hipMemcpyHostToDevice);
	

	dim3 BlockSize= ceil(K.nRow/256.00);
	dim3 ThreadSize = 256;

	cout<<"Entering the get D kernel..."<<endl;

	GetDiaKernel<<<BlockSize, ThreadSize>>>(DevA, DevD);
	hipMemcpy(D,DevD,sizeof(float)*(A.nRow),hipMemcpyDeviceToHost);

/*	for(int i = 0 ; i < A.nRow ; i++)
		cout<<D[i]<<" ";
	cout<<endl;*/

	//Get K from A
	cout<<"Entering the get K kernel..."<<endl;
	DLMulKernel<<<BlockSize,ThreadSize>>>(DevK, DevA, DevD);

	/*
	hipMemcpy(K.mData,DevK.mData,sizeof(float)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);
	hipMemcpy(K.mIndex,DevK.mIndex,sizeof(int)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);
	
	PrintELLFormat(K);
	*/
	hipMalloc((void **)&DevM.mData,sizeof(double)*A.nRow*A.nCol);
	hipMalloc((void **)&DevM.mIndex,sizeof(int)*A.nRow*A.nCol);



	dim3 BlockSize1= ceil(A.nRow/8.0);
	dim3 ThreadSize1 = 256;
	//Get M from K
	cout<<"Entering the get M kernel..."<<endl;
	ELLMulKernel<<<BlockSize1,ThreadSize1>>>(DevK,DevM,DevA);

	hipMemcpy(M.mData,DevM.mData,sizeof(float)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);
	hipMemcpy(M.mIndex,DevM.mIndex,sizeof(int)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime,start,stop);

	cout<<"the cost time is "<<elapseTime<<" ms "<<endl;

	hipFree(DevK.mData);
	hipFree(DevK.mIndex);

	hipFree(DevA.mData);
	hipFree(DevA.mIndex);

	hipFree(DevM.mData);
	hipFree(DevM.mIndex);

	hipFree(DevD);

	//cout<<"=====================ELL M Matrix Format==========================="<<endl;
	//PrintELLFormat(M);

	delete [] A.mData;
	delete [] A.mIndex;

	delete [] K.mData;
	delete [] K.mIndex;

	delete [] M.mData;
	delete [] M.mIndex;

	delete [] D;

	return elapseTime;

}

__global__ void GetDiaKernel(ELL_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;
	int Col;

	if(Row < DevA.nRow)
	{
		D[Row] = 0.0;

		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			Col = DevA.mIndex[i*DevA.nRow+Row];
			if(Row == Col)
			{
				D[Row] = DevA.mData[i*DevA.nRow+Row];
				break;
			}
		}
	}
}
__global__ void DLMulKernel(ELL_Matrix DevK, ELL_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;

	if(Row < DevA.nRow)
	{
		

		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			int Col = DevA.mIndex[i*DevA.nRow+Row];

			if(Row > Col)
			{
				DevK.mData[i*DevA.nRow+Row] = -(DevA.mData[i*DevA.nRow+Row]*(1/sqrt(D[Row]))*
					(1/D[Col]));
			}
			else if(Row == Col)
			{
				DevK.mData[i*DevA.nRow+Row] = (1/sqrt(D[Row]));
			}
			else
				DevK.mData[i*DevA.nRow+Row] = 0.0;

			DevK.mIndex[i*DevA.nRow+Row] = DevA.mIndex[i*DevA.nRow+Row];
		}
		
	}
}
__global__ void ELLMulKernel(ELL_Matrix DevK, ELL_Matrix DevM, ELL_Matrix DevA)
{
	__shared__ int RowIndex[8][1024]; 
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int warp_id = thread_id / 32;
	int lane = thread_id & (32 -1);

	int block_warp_id = warp_id %8;

	int Row = warp_id ;

	if(Row < DevA.nRow)
	{
		float dot = 0.0;

		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			RowIndex[block_warp_id][i] = DevK.mIndex[i*DevK.nRow+Row];
		}

		for(int l = lane ; l < DevA.nCol ; l+=32)
		{
			int Col = DevA.mIndex[l*DevK.nRow+Row];

			for(int i = 0 ; i < DevK.nCol ; i++)
			{
				int ColB = DevK.mIndex[i*DevK.nRow+Col];
			

				for(int j = 0 ; j < DevK.nCol ; j++)
				{
					int ColA = RowIndex[block_warp_id][j];
				
					if(ColA == ColB)
					{
						float valB =  DevK.mData[i*DevK.nRow+Col]; 
						float valA =  DevK.mData[j*DevK.nRow+Row];
							dot += valA*valB;
					}else if(ColB < ColA)
						break;
				}
			}
			DevM.mData[l*DevK.nRow+Row] += dot;
			DevM.mIndex[l*DevK.nRow+Row] = DevA.mIndex[l*DevK.nRow+Row];
		}
		
		//ELL*ELL
		
	}

}



