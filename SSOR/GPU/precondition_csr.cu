#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <string>
#include <math.h>
#include <assert.h>
#include "TiMatrix.h"
#include "TiMatrixType.h"

using namespace std;

inline void CUDA_CALL(hipError_t x)
{
	const hipError_t a = x;
	if(a != hipSuccess)
		cout<<"cuda error:"<<hipGetErrorString(a)<<endl;

	//hipDeviceReset();
	//assert(0);
}

__global__ void GetDiaKernel(CSR_Matrix DevA, float *D);
__global__ void DLMulKernel(CSR_Matrix DevK, CSR_Matrix DevA, float *D);
__global__ void CSRMulKernel(CSR_Matrix DevK, CSR_Matrix DevM, CSR_Matrix DevA);

float CSRMul(string fileName);


int main()
{
	float totaltime = 0;
	string filename;

	cout<<"please input matrix market file name:";
	cin>>filename;

	for(int i = 0 ; i < 5 ; i++)
	{
		totaltime += CSRMul(filename);
	}

	cout<<"the average cost time is "<<totaltime/5<<" ms "<<endl;

	return 0;

}



//����device memory������
float CSRMul(string fileName)
{
	
	
	CSR_Matrix A;
	CSR_Matrix K;
	CSR_Matrix M;
	
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	ReadMatrixMarketFile(fileName.c_str(),A);
	cout<<"Reading matrix..."<<endl;

	//print the matrix
	cout<<A.n<<"    "<<A.n<<"    "<<A.nonzeroes<<endl;
	//cout<<"=================The Matrix A Format======================"<<endl;
	//PrintCSRFormat(A);

	
	
	K.n = A.n;
	K.nonzeroes = A.nonzeroes;
	K.mData = new float[A.nonzeroes];
	K.mIndex = new int[A.nonzeroes];
	K.mPtr = new int[A.n+1];


	M.n = A.n;
	M.nonzeroes = A.nonzeroes;
	M.mData = new float[A.nonzeroes];
	M.mIndex = new int[A.nonzeroes];
	M.mPtr = new int[A.n+1];

	CSR_Matrix DevK;
	CSR_Matrix DevM;
	CSR_Matrix DevA;
	float *DevD;

	float *D = new float[A.n];


	DevA.n = A.n;
	DevA.nonzeroes = A.nonzeroes;

	
	DevK.n = K.n;
	DevK.nonzeroes = K.nonzeroes;

	DevM.n = M.n;
	DevM.nonzeroes = M.nonzeroes;


	hipMalloc((void **)&DevK.mData,sizeof(float)*A.nonzeroes);
	hipMalloc((void **)&DevK.mIndex,sizeof(int)*A.nonzeroes);
	hipMalloc((void **)&DevK.mPtr,sizeof(int)*(A.n+1));

	hipMalloc((void **)&DevA.mData,sizeof(float)*A.nonzeroes);
	hipMalloc((void **)&DevA.mIndex,sizeof(int)*A.nonzeroes);
	hipMalloc((void **)&DevA.mPtr,sizeof(int)*(A.n+1));

	hipMalloc((void **)&DevD, sizeof(float)*A.n);


	hipMemcpy(DevA.mData,A.mData,sizeof(float)*A.nonzeroes,hipMemcpyHostToDevice);
	hipMemcpy(DevA.mIndex,A.mIndex,sizeof(int)*A.nonzeroes,hipMemcpyHostToDevice);
	hipMemcpy(DevA.mPtr,A.mPtr,sizeof(int)*(A.n+1),hipMemcpyHostToDevice);

	hipMemcpy(DevK.mPtr,A.mPtr,sizeof(int)*(A.n+1),hipMemcpyHostToDevice);
	

	dim3 BlockSize = ceil(K.n/256.00);
	dim3 ThreadSize = 256;

	//Get the D from A
	//===================================

	cout<<"Entering the get D matrix kernel..."<<endl;
	GetDiaKernel<<<BlockSize,ThreadSize>>>(DevA, DevD);

	printf("%s\n",hipGetErrorString(hipGetLastError()));
	CUDA_CALL(hipMemcpy(D,DevD,sizeof(float)*(A.n),hipMemcpyDeviceToHost));
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	//cout<<"=================The Matrix D Format======================"<<endl;
	/*for(int i = 0 ; i < A.n ; i++)
		cout<<D[i]<<"    ";
	cout<<endl;*/
	


	//Get K from A
	cout<<"Entering the get K matrix kernel..."<<endl;
	//===================================
	DLMulKernel<<<BlockSize,ThreadSize>>>(DevK, DevA, DevD);

	printf("%s\n",hipGetErrorString(hipGetLastError()));

	hipMemcpy(K.mData,DevK.mData,sizeof(float)*(A.nonzeroes),hipMemcpyDeviceToHost);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(K.mIndex,DevK.mIndex,sizeof(int)*(A.nonzeroes),hipMemcpyDeviceToHost);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(K.mPtr,DevK.mPtr,sizeof(int)*(A.n+1),hipMemcpyDeviceToHost);
	printf("%s\n",hipGetErrorString(hipGetLastError()));

	//cout<<"=================The Matrix K Format======================"<<endl;
	//PrintCSRFormat(K);


	hipMalloc((void **)&DevM.mData,sizeof(float)*A.nonzeroes);
	hipMalloc((void **)&DevM.mIndex,sizeof(int)*A.nonzeroes);
	hipMalloc((void **)&DevM.mPtr,sizeof(int)*(A.n+1));

	hipMemcpy(DevM.mPtr,A.mPtr,sizeof(int)*(A.n+1),hipMemcpyHostToDevice);

	dim3 BlockSize1= ceil(A.n/8.0);
	dim3 ThreadSize1 = 256;


	//Get M from K
	//===================================
	cout<<"Entering the get M matrix kernel..."<<endl;
	CSRMulKernel<<<BlockSize1,ThreadSize1>>>(DevK,DevM,DevA);
	printf("%s\n",hipGetErrorString(hipGetLastError()));

	hipMemcpy(M.mData,DevM.mData,sizeof(float)*(M.nonzeroes),hipMemcpyDeviceToHost);
	hipMemcpy(M.mIndex,DevM.mIndex,sizeof(int)*(M.nonzeroes),hipMemcpyDeviceToHost);
	hipMemcpy(M.mPtr,DevM.mPtr,sizeof(int)*(M.n+1),hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime,start,stop);

	hipFree(DevK.mData);
	hipFree(DevK.mIndex);
	hipFree(DevK.mPtr);

	hipFree(DevA.mData);
	hipFree(DevA.mIndex);
	hipFree(DevA.mPtr);

	hipFree(DevM.mData);
	hipFree(DevM.mIndex);
	hipFree(DevM.mPtr);

	//cout<<"=================The Matrix M Format======================"<<endl;
	//PrintCSRFormat(M);
	delete [] A.mData;
	delete [] A.mIndex;
	delete [] A.mPtr;

	delete [] K.mData;
	delete [] K.mIndex;
	delete [] K.mPtr;

	delete [] M.mData;
	delete [] M.mIndex;
	delete [] M.mPtr;

	

	cout<<"the time cost is "<<elapseTime<<" ms."<<endl;
	return elapseTime;

}

__global__ void GetDiaKernel(CSR_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;
	int begin,end;

	if(Row < DevA.n)
	{

		begin = DevA.mPtr[Row];
		end = DevA.mPtr[Row+1];

		D[Row] = 0.0;

		
		for(int i = begin ; i < end ; i++)
		{
			if(Row == DevA.mIndex[i])
			{
				D[Row] = DevA.mData[i];
			}
		}

	}
	
}

__global__ void DLMulKernel(CSR_Matrix DevK, CSR_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;

	int begin,end;

	if(Row < DevA.n)
	{

		end = DevA.mPtr[Row+1];
		begin = DevA.mPtr[Row];

		for(int i = begin ; i < end ; i++)
		{
			int Col = DevA.mIndex[i];

			if(Row > Col)
			{
				DevK.mData[i] = -(DevA.mData[i]*(1/sqrt(D[Row]))*
					(1/D[Col]));
			}
			else if(Row == Col)
			{
				DevK.mData[i] = (1/sqrt(D[Row]));
			}
			else
				DevK.mData[i] = 0.0;

			DevK.mIndex[i] = DevA.mIndex[i];
		}
	}
}

__global__ void CSRMulKernel(CSR_Matrix DevK, CSR_Matrix DevM, CSR_Matrix DevA)
{
	//SpMM kernel for the CSR sparse matrix format using one 32-thread warp per matrix row of M

	__shared__ int RowIndex[8][1024]; 
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int warp_id = thread_id / 32;
	int lane = thread_id & (32 -1);

	int block_warp_id = warp_id %8;

	int Row = warp_id ;
	
	if(Row < DevA.n)
	{
		float dot = 0.0;
		int RowBegin = DevK.mPtr[Row];
		int RowEnd = DevK.mPtr[Row+1];

		for(int i = RowBegin ; i < RowEnd ; i++)
		{
			RowIndex[block_warp_id][i - RowBegin] = DevK.mIndex[i];
		}

		int beginA = DevA.mPtr[Row];
		int endA = DevA.mPtr[Row+1];

		for(int j = beginA + lane ; j < endA ; j += 32)
		{
			int Col = DevA.mIndex[j];

			int ColBegin = DevK.mPtr[Col];
			int ColEnd = DevK.mPtr[Col+1];

			for(int k = ColBegin ; k < ColEnd ; k++)
			{
				for(int l = RowBegin ; l < RowEnd ; l++)
				{
					if(DevK.mIndex[k] == RowIndex[block_warp_id][l-RowBegin])
					{
						dot += DevK.mData[k]*DevK.mData[l];
						break;
					}
				}
			}

			DevM.mData[j] += dot;
			DevM.mIndex[j] = DevA.mIndex[j];
		}
	}
}


