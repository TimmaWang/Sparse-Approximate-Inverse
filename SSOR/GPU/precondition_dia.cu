#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <iostream>
#include <string>
#include <math.h>
#include "TiMatrix.h"
#include "TiMatrixType.h"

using namespace std;

inline void CUDA_CALL(hipError_t x)
{
	const hipError_t a = x;
	if(a != hipSuccess)
		cout<<"cuda error:"<<hipGetErrorString(a)<<endl;

	//hipDeviceReset();
	//assert(0);
}


__global__ void GetDiaKernel(DIA_Matrix DevA, float *D);
__global__ void DLMulKernel(DIA_Matrix DevK, DIA_Matrix DevA, float *D);
__global__ void DIAMulKernel(DIA_Matrix DevK, DIA_Matrix DevM, DIA_Matrix DevA);

float DIAMul(string filename);


int main()
{
	float totaltime = 0;
	string filename;

	cout<<"please input matrix market file name:";
	cin>>filename;

	for(int i = 0 ; i < 5 ; i++)
		totaltime += DIAMul(filename);

	cout<<"the average cost time is "<<totaltime/5<<" ms "<<endl;

	return 0;

}



//����device memory������
float DIAMul(string filename)
{
	
	

	CSR_Matrix origin;
	DIA_Matrix A;
	DIA_Matrix K;
	DIA_Matrix M;
	float *D;
	
	ReadMatrixMarketFile(filename.c_str(),origin);
	cout<<"Reading matrix..."<<endl;

	CSR2DIA(origin, A);

	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	//print the matrix
	cout<<A.nRow<<"    "<<A.nCol<<"    "<<A.nonzeroes<<endl;
	//PrintDIAFormat(A);

	K.nRow = A.nRow;
	K.nCol = A.nCol;
	K.nonzeroes = A.nonzeroes;

	K.mData = new float[A.nRow*A.nCol];
	K.mOffset = new int[A.nCol];


	M.nRow = A.nRow;
	M.nCol = A.nCol;
	M.nDIG = A.nDIG;
	M.nonzeroes = A.nonzeroes;
	M.mData = new float[A.nRow*A.nCol];
	M.mOffset = new int[A.nCol];

	D = new float[A.nRow];

	DIA_Matrix DevK;
	DIA_Matrix DevM;
	DIA_Matrix DevA;
	float *DevD;


	DevA.nRow = A.nRow;
	DevA.nCol = A.nCol;
	DevA.nDIG = A.nDIG;
	DevA.nonzeroes = A.nonzeroes;

	DevK.nRow = K.nRow;
	DevK.nCol = K.nCol;
	DevK.nonzeroes = K.nonzeroes;

	DevM.nRow = M.nRow;
	DevM.nRow = M.nCol;
	DevM.nonzeroes = M.nonzeroes;


	hipMalloc((void **)&DevK.mData,sizeof(float)*A.nRow*A.nCol);
	hipMalloc((void **)&DevK.mOffset,sizeof(int)*A.nCol);

	hipMalloc((void **)&DevA.mData,sizeof(float)*A.nRow*A.nCol);
	hipMalloc((void **)&DevA.mOffset,sizeof(int)*A.nCol);

	hipMalloc((void **)&DevD, sizeof(float)*A.nRow);


	hipMemcpy(DevA.mData,A.mData,sizeof(float)*A.nRow*A.nCol,hipMemcpyHostToDevice);
	hipMemcpy(DevA.mOffset,A.mOffset,sizeof(int)*A.nCol,hipMemcpyHostToDevice);

	hipMemcpy(DevK.mOffset,A.mOffset,sizeof(int)*A.nCol,hipMemcpyHostToDevice);
	

	dim3 BlockSize= ceil(K.nRow/256.00);
	dim3 ThreadSize = 256;

	cout<<"Entering the get D kernel..."<<endl;

	GetDiaKernel<<<BlockSize, ThreadSize>>>(DevA, DevD);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(D,DevD,sizeof(float)*(A.nRow),hipMemcpyDeviceToHost);

	/*for(int i = 0 ; i < A.nRow ; i++)
		cout<<D[i]<<" ";
	cout<<endl;*/

	//Get K from A
	cout<<"Entering the get K kernel..."<<endl;
	DLMulKernel<<<BlockSize,ThreadSize>>>(DevK, DevA, DevD);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	
	//cout<<"=======================K Matirx===================="<<endl;
	hipMemcpy(K.mData,DevK.mData,sizeof(float)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);
	hipMemcpy(K.mOffset,DevK.mOffset,sizeof(int)*(A.nCol),hipMemcpyDeviceToHost);
	//PrintDIAFormat(K);


	hipMalloc((void **)&DevM.mData,sizeof(float)*A.nRow*A.nCol);
	hipMalloc((void **)&DevM.mOffset,sizeof(int)*A.nCol);

	dim3 BlockSize1= ceil(A.nRow/8.0);
	dim3 ThreadSize1 = 256;
	//Get M from K
	cout<<"Entering the get M kernel..."<<endl;
	DIAMulKernel<<<BlockSize1,ThreadSize1>>>(DevK,DevM,DevA);
	printf("%s\n",hipGetErrorString(hipGetLastError()));
	hipMemcpy(M.mData,DevM.mData,sizeof(float)*(A.nRow*A.nCol),hipMemcpyDeviceToHost);
	hipMemcpy(M.mOffset,DevM.mOffset,sizeof(int)*(A.nCol),hipMemcpyDeviceToHost);

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime,start,stop);

	cout<<"the cost time is "<<elapseTime<<" ms "<<endl;

	hipFree(DevK.mData);
	hipFree(DevK.mOffset);

	hipFree(DevA.mData);
	hipFree(DevA.mOffset);

	hipFree(DevM.mData);
	hipFree(DevM.mOffset);

	hipFree(DevD);

	//cout<<"=======================M Matirx===================="<<endl;
	//PrintDIAFormat(M);

	delete [] A.mData;
	delete [] A.mOffset;

	delete [] K.mData;
	delete [] K.mOffset;

	delete [] M.mData;
	delete [] M.mOffset;


	return elapseTime;

}

__global__ void GetDiaKernel(DIA_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;
	int Col;

	if(Row < DevA.nRow)
	{
		D[Row] = 0.0;

		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			Col = Row + DevA.mOffset[i];
			if(Row == Col)
			{
				D[Row] = DevA.mData[i*DevA.nRow+Row];
				break;
			}
		}
	}
}
__global__ void DLMulKernel(DIA_Matrix DevK, DIA_Matrix DevA, float *D)
{
	int Row = blockDim.x * blockIdx.x+ threadIdx.x;

	if(Row < DevA.nRow)
	{
		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			int Col = Row + DevA.mOffset[i];

			if(Col >= 0 && Col < DevA.nRow)
			{
				if(Row > Col)
				{
					DevK.mData[i*DevA.nRow+Row] = -(DevA.mData[i*DevA.nRow+Row]*(1/sqrt(D[Row]))*
						(1/D[Col]));
				}
				else if(Row == Col)
				{
					DevK.mData[i*DevA.nRow+Row] = (1/sqrt(D[Row]));
				}
				else
					DevK.mData[i*DevA.nRow+Row] = 0.0;

			}
		}
		
	}
}

__global__ void DIAMulKernel(DIA_Matrix DevK, DIA_Matrix DevM, DIA_Matrix DevA)
{
	__shared__ int RowIndex[8][1024]; 
	int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
	int warp_id = thread_id / 32;
	int lane = thread_id & (32 -1);

	int block_warp_id = warp_id %8;

	int Row = warp_id ;

	if(Row < DevA.nRow)
	{

		float dot = 0.0;

		for(int i = 0 ; i < DevA.nCol ; i++)
		{
			RowIndex[block_warp_id][i] = Row + DevK.mOffset[i];
		}
		
		for(int l = lane ; l < DevA.nCol ; l+=32)
		{
			int Col = Row + DevA.mOffset[l];

			for(int i = 0 ; i < DevK.nCol ; i++)
			{
				int ColB = Col + DevK.mOffset[i];
			

				for(int j = 0 ; j < DevK.nCol ; j++)
				{
					int ColA = RowIndex[block_warp_id][j];
				
					if(ColA == ColB)
					{
						float valB =  DevK.mData[i*DevK.nRow+Col]; 
						float valA =  DevK.mData[j*DevK.nRow+Row];
							dot += valA*valB;
					}else if(ColB < ColA)
						break;
				}
			}
			DevM.mData[l*DevK.nRow+Row] += dot;
			DevM.mOffset[l] = DevA.mOffset[l];
		}

		//DIA * DIA 
	}
}



